#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

__global__ void SomeTransform(char *input_gpu, int fsize) {
	// thread ID 一律從 1 開始
	int idx = threadIdx.x;
	int i;
	/// 32 thread 同時輪流執行
	for( i = idx; i < fsize; i += 32 ){
		// 將全部字母修改為大寫英文字母A
		if( input_gpu[i] != '\n' ){
			input_gpu[i] = 'A';
		}
	}
}

int main(int argc, char **argv)
{
	// init, and check
	if (argc != 2) {
		printf("Usage %s <input text file>\n", argv[0]);
		abort();
	}
	FILE *fp = fopen(argv[1], "r");
	if (not fp) {
		printf("Cannot open %s", argv[1]);
		abort();
	}
	// get file size
	fseek(fp, 0, SEEK_END);
	size_t fsize = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	// read files
	MemoryBuffer<char> text(fsize+1);
	auto text_smem = text.CreateSync(fsize);
	CHECK;
	fread(text_smem.get_cpu_wo(), 1, fsize, fp);
	text_smem.get_cpu_wo()[fsize] = '\0';
	fclose(fp);

	// TODO: do your transform here
	char *input_gpu = text_smem.get_gpu_rw();
	// 用 1 個block 和 32個thread 來處理字元
	SomeTransform<<<1, 32>>>(input_gpu, fsize);

	puts(text_smem.get_cpu_ro());
	return 0;
}




